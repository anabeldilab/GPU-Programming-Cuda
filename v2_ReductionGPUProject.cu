/*
 ============================================================================
 Proyecto GPU
 Realizado por: Anabel Díaz Labrador

 ============================================================================
 */


#include <iostream>

#include <hip/hip_runtime.h>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

void show_vector(int*, int, int);

#define N (1048576)
#define M (8)
#define HILOSPORBLOQUE (512)


// Device kernel
__global__ void incHist(const int *A, int numElements, int *histogram, int numElementsHistograms) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (i < numElements) {
    int increment = A[i] % M;
    atomicAdd(&histogram[M * blockIdx.x + increment], 1);
  }
}

// Device kernel
__global__ void reduccion_paralela(int *histogram, int numElements, int *result) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    for(unsigned int middle = numElements / 2; middle >= M; middle >>= 1) { // We reduce until there are 8 left to join. Last iteration middle = 4
      if (i < middle) {
        histogram[i] = histogram[i] + histogram[i + middle];
      }
      __syncthreads();
    }
  } 

  if (i >= 0 && i < M) {
    result[i] = histogram[i];
  }
}


// Host main
int main(void) {
  // Vector length to be used, and compute its size
  const int numElementsA = N;
  size_t sizeA = numElementsA * sizeof(int);

  // Allocate the host input vector A
  int *h_A = (int*)malloc(sizeA);

  // Verify that allocations succeeded
  if (h_A == NULL) {
      fprintf(stderr, "Failed to allocate host vectors!\n");
      exit(EXIT_FAILURE);
  }

  // Initialize the host input vector with [0, 1000000] random range
  time_t t;
  srand((unsigned) time(&t));
  for (int i = 0; i < numElementsA; i++) {
    h_A[i] = rand() % N;
  }
  printf("Vector element number: %d\n", numElementsA);
  //show_vector(h_A, 0, 10); Check that adds random numbers

  // Allocate the device input vector A
  int *d_A = NULL;
  CUDA_CHECK_RETURN(hipMalloc((void**)&d_A, sizeA));

  // Copy the host input vector A in host memory to the device input vector in
  // device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  CUDA_CHECK_RETURN(hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice));

  // Calculate the number of blocks needed
  int threadsPerBlock = HILOSPORBLOQUE;
  int blocksPerGrid = (numElementsA + threadsPerBlock - 1) / threadsPerBlock;

  // Vector length to be used, and compute its size
  int numElementsHistograms = blocksPerGrid * M;
  size_t sizeHistograms = numElementsHistograms * sizeof(int);

  // Allocate the host input vector histogram
  int *h_histograms = (int*)malloc(sizeHistograms);
  if (h_histograms == NULL) {
      fprintf(stderr, "Failed to allocate host vectors!\n");
      exit(EXIT_FAILURE);
  }

  // Initialize the host input vector
  for (int i = 0; i < numElementsHistograms; i++) {
    h_histograms[i] = 0;
  }

  // Allocate the device input vector histogram
  int *d_histograms = NULL;
  CUDA_CHECK_RETURN(hipMalloc((void**)&d_histograms, sizeHistograms));

  // Copy the host input vector histograms in host memory to the device input vector in
  // device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  CUDA_CHECK_RETURN(hipMemcpy(d_histograms, h_histograms, sizeHistograms, hipMemcpyHostToDevice));

  // Launch the incHist CUDA Kernel
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  incHist<<<blocksPerGrid, threadsPerBlock>>>(d_A, numElementsA, d_histograms, numElementsHistograms);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  CUDA_CHECK_RETURN(hipGetLastError());

  float elapsedTime1;
  hipEventElapsedTime(&elapsedTime1, start, stop);
  
  // Get back the histograms
  printf("Copy local histograms from the CUDA device to the host memory\n");
  CUDA_CHECK_RETURN(hipMemcpy(h_histograms, d_histograms, sizeHistograms, hipMemcpyDeviceToHost));

  // Vector check
  printf("\nFirst 8 elements of the vector of local histograms: ");
  show_vector(h_histograms, 0, M);
  int acc = 0;
  for (int i = 0; i < numElementsHistograms; i++) {
    acc += h_histograms[i];
  }
  printf("Histogram total increments: %d\nHistogram size: %d\n", acc, numElementsHistograms);

  // Vector length to be used, and compute its size
  int numElementsHistogram = M;
  size_t sizeHistogram = numElementsHistogram * sizeof(int);


  // Allocate the host input vector histograma
  int *h_histogram = (int*)malloc(sizeHistogram);
  if (h_histogram == NULL) {
      fprintf(stderr, "Failed to allocate host vectors!\n");
      exit(EXIT_FAILURE);
  }

  // Allocate the device input vector histogram
  int *d_histogram = NULL;
  CUDA_CHECK_RETURN(hipMalloc((void**)&d_histogram, sizeHistogram));

  // Initialize the host input vector
  for (int i = 0; i < M; i++) {
    h_histogram[i] = 0;
  }

  // Copy the host input vector histogram in host memory to the device input vector in
  // device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  CUDA_CHECK_RETURN(hipMemcpy(d_histogram, h_histogram,  sizeHistogram, hipMemcpyHostToDevice));

  // Launch the reduccion_paralela CUDA Kernel
  blocksPerGrid = (numElementsHistograms + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  hipEventRecord(start, 0);
  reduccion_paralela<<<blocksPerGrid, threadsPerBlock>>>(d_histograms, numElementsHistograms, d_histogram);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  CUDA_CHECK_RETURN(hipGetLastError());

  float elapsedTime2;
  hipEventElapsedTime(&elapsedTime2, start, stop);

  // Get back the histogram result
  printf("Copy histogram result from the CUDA device to the host memory\n");
  CUDA_CHECK_RETURN(hipMemcpy(h_histogram, d_histogram, sizeHistogram, hipMemcpyDeviceToHost));

  // Vector check
  show_vector(h_histogram, 0, M);
  acc = 0;
  for (int i = 0; i < M; i++) {
    acc += h_histogram[i];
  }
  printf("Histogram total data: %d\n", acc);

  // Free device global memory
  CUDA_CHECK_RETURN(hipFree(d_A));
  CUDA_CHECK_RETURN(hipFree(d_histograms));
  CUDA_CHECK_RETURN(hipFree(d_histogram));

  // Free host memory
  free(h_A);
  free(h_histograms);
  free(h_histogram);

  printf("\nTiempo construyendo histogramas locales: %f milisegundos\n", elapsedTime1);
  printf("Tiempo juntando histogramas en uno final: %f milisegundos\n", elapsedTime2);
  printf("Tiempo total: %f milisegundos\n", elapsedTime1 + elapsedTime2);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  printf("Done\n");
  return EXIT_SUCCESS;
}


// Check the return value of the CUDA runtime API call and exit the application if the call has failed.
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err) {

	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (EXIT_FAILURE);
}


// Returns a range given the vector by the terminal
void show_vector(int* vector, int min, int max) {
  printf("[%d", vector[min]);
  for (unsigned i = min + 1; i < max; i++) 
    printf(", %d", vector[i]);   
  printf("]\n");
}